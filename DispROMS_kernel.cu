#include "hip/hip_runtime.h"
// declare texture reference for 2D float texture

texture<float, 2, hipReadModeElementType> texU;
texture<float, 2, hipReadModeElementType> texV;
texture<float, 2, hipReadModeElementType> texlonu;
texture<float, 2, hipReadModeElementType> texlatu;
texture<float, 2, hipReadModeElementType> texdXU;
texture<float, 2, hipReadModeElementType> texdYV;



__global__ void HD_interp(int nnode,int stp,int backswitch, int nhdstp, float dt, float hddt/*,float *Umask*/,float * Uold,float * Unew, float * UU)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tx =threadIdx.x;

	__shared__ float Uxo[16];
	__shared__ float Uxn[16];
	__shared__ float Ums[16];
	
	
	float fac=1.0;
	/*Ums[tx]=Umask[ix];*/
	
	
	if (backswitch>0)
	{
		fac=-1.0f;
	}
	
	
	if (ix<nnode)
	{
		Uxo[tx]=fac*Uold[ix]/**Ums[tx]*/;
		Uxn[tx]=fac*Unew[ix]/**Ums[tx]*/;
		
		UU[ix]=Uxo[tx]+(stp*dt-hddt*nhdstp)*(Uxn[tx]-Uxo[tx])/hddt;
	}
}


__global__ void NextHDstep(int nnode, float * Uold,float * Unew)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	
	
	if (ix<nnode )
	{
		Uold[ix]=Unew[ix];
	}
}

__global__ void updatepartpos(int npart,float dt,float Eh,float * dd_rand,float *xx, float *yy,float *zz,float *tt)	 
{
	int i = blockIdx.x * blockDim.x * blockDim.y + blockDim.x * threadIdx.y + threadIdx.x;
	
	float Ux=0.0f;
	float Vx=0.0f;
	
	float distu, distv;
	
	
	float xxx,yyy,ttt;
	xxx=xx[i];
	yyy=yy[i];
	ttt=tt[i];
		
	if(ttt>=0.0f)
	{
	//Interpolate wter depth, Uvel Vvel at the particle position
     
    Ux=tex2D(texU, xxx, yyy);
    Vx=tex2D(texV, xxx+0.5, yyy-0.5);// U and V don't have the same coordinates but in the number of nodes it is just off by half a grid node in both dimension
    distu=tex2D(texdXU, xxx, yyy);
    distv=tex2D(texdYV, xxx+0.5, yyy-0.5);
    
  	xx[i]=xxx+(Ux*dt+(dd_rand[i]*2-1)*sqrtf(6*Eh*dt))/distu;
	yy[i]=yyy+(Vx*dt+(dd_rand[npart-i]*2-1)*sqrtf(6*Eh*dt))/distv;
	}
	tt[i]=ttt+dt;
	
    
}

__global__ void ij2lonlat(int npart, float * xx, float *yy, float *xp, float *yp)
{
	int i = blockIdx.x * blockDim.x * blockDim.y + blockDim.x * threadIdx.y + threadIdx.x;
	float lon;
	float lat;
	float xxx,yyy;
	xxx=xx[i];
	yyy=yy[i];
	
	lon=tex2D(texlonu, xxx, yyy);
    lat=tex2D(texlatu, xxx, yyy);
    
    xp[i]=lon;
    yp[i]=lat;
	
	//
	
} 
